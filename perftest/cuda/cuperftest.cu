#include <getopt.h>

#include <cstdlib>
#include <iostream>
#include <random>
#include <string>
#include <unordered_map>

#include <cufinufft.h>
#include <cufinufft/impl.h>

#include <thrust/complex.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

std::string get_or(const std::unordered_map<std::string, std::string> &m, const std::string &key,
                   const std::string &default_value) {
    auto it = m.find(key);
    if (it == m.end()) {
        return default_value;
    }
    return it->second;
}

struct test_options_t {
    char prec;
    int type;
    int n_runs;
    int N[3];
    int M;
    int ntransf;
    int kerevalmethod;
    int method;
    int sort;
    double tol;

    test_options_t(int argc, char *argv[]) {
        std::unordered_map<std::string, std::string> options_map;

        while (true) {
            int option_index = 0;

            // clang-format off
            static struct option long_options[] {
                {"prec", required_argument, 0, 0},
                {"type", required_argument, 0, 0},
                {"n_runs", required_argument, 0, 0},
                {"N1", required_argument, 0, 0},
                {"N2", required_argument, 0, 0},
                {"N3", required_argument, 0, 0},
                {"M", required_argument, 0, 0},
                {"ntransf", required_argument, 0, 0},
                {"tol", required_argument, 0, 0},
                {"method", required_argument, 0, 0},
                {"kerevalmethod", required_argument, 0, 0},
                {"sort", required_argument, 0, 0},
                {0, 0, 0, 0},
            };
            // clang-format on

            int c = getopt_long(argc, argv, "", long_options, &option_index);
            if (c == -1)
                break;

            switch (c) {
            case 0:
                options_map[long_options[option_index].name] = optarg;
                break;

            default:
                break;
            }
        }

        prec = get_or(options_map, "prec", "f")[0];
        type = std::stoi(get_or(options_map, "type", "1"));
        n_runs = std::stoi(get_or(options_map, "n_runs", "10"));
        N[0] = std::stof(get_or(options_map, "N1", "1E6"));
        N[1] = std::stof(get_or(options_map, "N2", "1"));
        N[2] = std::stof(get_or(options_map, "N3", "1"));
        M = std::stof(get_or(options_map, "M", "2E6"));
        ntransf = std::stoi(get_or(options_map, "ntransf", "1"));
        method = std::stoi(get_or(options_map, "method", "1"));
        kerevalmethod = std::stoi(get_or(options_map, "kerevalmethod", "1"));
        sort = std::stoi(get_or(options_map, "sort", "1"));
        tol = std::stof(get_or(options_map, "tol", "1E-5"));
    }

    friend std::ostream &operator<<(std::ostream &outs, const test_options_t &opts) {
        return outs << "prec = " << opts.prec << "\n"
                    << "type = " << opts.type << "\n"
                    << "n_runs = " << opts.n_runs << "\n"
                    << "N1 = " << opts.N[0] << "\n"
                    << "N2 = " << opts.N[1] << "\n"
                    << "N3 = " << opts.N[2] << "\n"
                    << "M = " << opts.M << "\n"
                    << "ntransf = " << opts.ntransf << "\n"
                    << "method = " << opts.method << "\n"
                    << "kerevalmethod = " << opts.kerevalmethod << "\n"
                    << "sort = " << opts.sort << "\n"
                    << "tol = " << opts.tol << "\n";
    }
};

struct CudaTimer {
    CudaTimer() {}

    ~CudaTimer() {
        for (auto &event : start_)
            hipEventDestroy(event);
        for (auto &event : stop_)
            hipEventDestroy(event);
    }

    void start() {
        start_.push_back(hipEvent_t{});
        stop_.push_back(hipEvent_t{});

        hipEventCreate(&start_.back());
        hipEventCreate(&stop_.back());

        hipEventRecord(start_.back());
    }

    void stop() { hipEventRecord(stop_.back()); }

    float elapsed() {
        float dt_tot = 0.;
        for (int i = 0; i < start_.size(); ++i) {
            float dt;
            hipEventSynchronize(stop_[i]);
            hipEventElapsedTime(&dt, start_[i], stop_[i]);
            dt_tot += dt;
        }

        return dt_tot;
    }

    std::vector<hipEvent_t> start_;
    std::vector<hipEvent_t> stop_;
};

template <class F, class... Args>
inline void timeit(F f, CudaTimer &timer, Args... args) {
    timer.start();
    f(args...);
    timer.stop();
}

void gpu_warmup() {
    int nf1 = 100;
    hipfftHandle fftplan;
    hipfftPlan1d(&fftplan, nf1, HIPFFT_Z2Z, 1);
    thrust::device_vector<hipfftDoubleComplex> in(nf1), out(nf1);
    hipfftExecZ2Z(fftplan, in.data().get(), out.data().get(), 1);
    hipDeviceSynchronize();
}

template <typename T>
void run_test(test_options_t &test_opts) {
    std::cout << test_opts;
    const int ntransf = test_opts.ntransf;
    const int M = test_opts.M;
    const int N = test_opts.N[0] * test_opts.N[1] * test_opts.N[2];
    const int type = test_opts.type;
    constexpr int iflag = 1;

    thrust::host_vector<T> x(M * ntransf), y(M * ntransf), z(M * ntransf);
    thrust::host_vector<thrust::complex<T>> c(M * ntransf), fk(N * ntransf);

    thrust::device_vector<T> d_x(M * ntransf), d_y(M * ntransf), d_z(M * ntransf);
    thrust::device_vector<thrust::complex<T>> d_c(M * ntransf), d_fk(N * ntransf);

    std::default_random_engine eng(1);
    std::uniform_real_distribution<T> dist11(-1, 1);
    auto randm11 = [&eng, &dist11]() { return dist11(eng); };

    // Making data
    for (int i = 0; i < M * ntransf; i++) {
        x[i] = M_PI * randm11(); // x in [-pi,pi)
        y[i] = M_PI * randm11();
        z[i] = M_PI * randm11();
    }
    d_x = x, d_y = y, d_z = z;

    if (type == 1) {
        for (int i = 0; i < M * ntransf; i++) {
            c[i].real(randm11());
            c[i].imag(randm11());
        }
        d_c = c;
    } else if (type == 2) {
        for (int i = 0; i < N * ntransf; i++) {
            fk[i].real(randm11());
            fk[i].imag(randm11());
        }
        d_fk = fk;
    } else {
        std::cerr << "Invalid type " << type << " supplied\n";
        return;
    }

    gpu_warmup();

    cufinufft_opts opts;
    int dim = 0;
    for (int i = 0; i < 3; ++i)
        dim = test_opts.N[i] > 1 ? i + 1 : dim;

    cufinufft_default_opts(test_opts.type, dim, &opts);
    opts.gpu_method = test_opts.method;
    opts.gpu_sort = test_opts.sort;
    opts.gpu_kerevalmeth = test_opts.kerevalmethod;

    cufinufft_plan_t<T> *dplan;
    CudaTimer makeplan_timer, setpts_timer, execute_timer;
    timeit(cufinufft_makeplan_impl<T>, makeplan_timer, test_opts.type, dim, test_opts.N, iflag, ntransf, test_opts.tol,
           &dplan, &opts);

    T *d_x_p = dim >= 1 ? d_x.data().get() : nullptr;
    T *d_y_p = dim >= 2 ? d_y.data().get() : nullptr;
    T *d_z_p = dim == 3 ? d_z.data().get() : nullptr;
    cuda_complex<T> *d_c_p = (cuda_complex<T> *)d_c.data().get();
    cuda_complex<T> *d_fk_p = (cuda_complex<T> *)d_fk.data().get();
    for (int i = 0; i < test_opts.n_runs; ++i) {
        timeit(cufinufft_setpts_impl<T>, setpts_timer, M, d_x_p, d_y_p, d_z_p, 0, nullptr, nullptr, nullptr, dplan);
        timeit(cufinufft_execute_impl<T>, execute_timer, d_c_p, d_fk_p, dplan);
    }

    float scale_factor = 1.0 / (test_opts.n_runs * ntransf);

    std::cout << std::endl;
    std::cout << "makeplan: " << makeplan_timer.elapsed() << " ms\n";
    std::cout << "setpts  : " << scale_factor * setpts_timer.elapsed() << " ms\n";
    std::cout << "execute : " << scale_factor * execute_timer.elapsed() << " ms\n";
    std::cout << "total   : "
              << makeplan_timer.elapsed() + scale_factor * (setpts_timer.elapsed() + execute_timer.elapsed())
              << " ms\n";
}

int main(int argc, char *argv[]) {
    if (argc == 2 && (std::string(argv[1]) == "--help" || std::string(argv[1]) == "-h")) {
        test_options_t default_opts(0, nullptr);
        // clang-format off
        std::cout << "Valid options:\n"
                     "    --prec <char>\n"
                     "           float or double precision. i.e. 'f' or 'd'\n"
                     "           default: " << default_opts.prec << "\n" << 
                     "    --type <int>\n"
                     "           type of transform. 1 or 2\n"
                     "           default: " << default_opts.type << "\n" << 
                     "    --n_runs <int>\n"
                     "           number of runs to average performance over\n"
                     "           default: " << default_opts.n_runs << "\n" <<
                     "    --N1 <int>\n"
                     "           number of modes in first dimension. Scientific notation accepted (i.e. 1E6)\n"
                     "           default: " << default_opts.N[0] << "\n" <<
                     "    --N2 <int>\n"
                     "           number of modes in second dimension. Scientific notation accepted (i.e. 1E6)\n"
                     "           default: " << default_opts.N[1] << "\n" <<
                     "    --N3 <int>\n"
                     "           number of modes in third dimension. Scientific notation accepted (i.e. 1E6)\n"
                     "           default: " << default_opts.N[2] << "\n" <<
                     "    --M <int>\n"
                     "           number of non-uniform points. Scientific notation accepted (i.e. 1E6)\n"
                     "           default: " << default_opts.M << "\n" <<
                     "    --ntransf <int>\n"
                     "           number of transforms to do simultaneously\n"
                     "           default: " << default_opts.ntransf << "\n" <<
                     "    --tol <float>\n"
                     "           NUFFT tolerance. Scientific notation accepted (i.e. 1.2E-7)\n"
                     "           default: " << default_opts.tol << "\n" <<
                     "    --method <int>\n"
                     "           NUFFT method\n"
                     "               1: nupts driven\n"
                     "               2: sub-problem\n"
                     "               4: block-gather\n"
                     "           Note that not all methods are compatible with all dim/type combinations\n"
                     "           default: " << default_opts.method << "\n" <<
                     "    --kerevalmeth <int>\n"
                     "           kernel evaluation method\n"
                     "               0: Exponential of square root\n"
                     "               1: Horner evaluation\n"
                     "           default: " << default_opts.kerevalmethod << "\n" <<
                     "    --sort: <int>\n"
                     "           sort strategy\n"
                     "               0: do not sort the points\n"
                     "               1: sort the points\n"
                     "           default: " << default_opts.sort << "\n";
        // clang-format on
        return 0;
    }
    test_options_t opts(argc, argv);

    if (opts.prec == 'f')
        run_test<float>(opts);
    else if (opts.prec == 'd')
        run_test<double>(opts);

    return 0;
}
